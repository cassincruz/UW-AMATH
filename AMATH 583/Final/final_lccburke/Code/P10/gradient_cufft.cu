#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <cmath>
#include <chrono>

__device__ hipDoubleComplex complexToCuComplex(double real, double imag)
{
    return make_hipDoubleComplex(real, imag);
}

__device__ void cuComplexToComplex(hipDoubleComplex c, double* real, double* imag)
{
    *real = hipCreal(c);
    *imag = hipCimag(c);
}

__device__ hipDoubleComplex operator*(hipDoubleComplex a, hipDoubleComplex b)
{
    return make_hipDoubleComplex(hipCreal(a) * hipCreal(b) - hipCimag(a) * hipCimag(b),
                                hipCreal(a) * hipCimag(b) + hipCimag(a) * hipCreal(b));
}

__device__ hipDoubleComplex operator/(hipDoubleComplex a, double b)
{
    return make_hipDoubleComplex(hipCreal(a) / b, hipCimag(a) / b);
}

__global__ void computeDdx(hipDoubleComplex* wave_gpu, hipDoubleComplex* fft_3_gpu, double kx, int nxyz)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nxyz) {
        hipDoubleComplex factor = make_hipDoubleComplex(0.0, kx);
        fft_3_gpu[idx] = wave_gpu[idx] * factor;
    }
}

__global__ void computeScale(hipDoubleComplex* fft_3_gpu, hipDoubleComplex* d_dx_gpu, double scale, int nxyz)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nxyz) {
        d_dx_gpu[idx] = fft_3_gpu[idx] / scale;
    }
}

int main()
{
    int ntrials = 3;

    // Allocating memory
    int nx, ny, nz, nxyz, ix, iy, iz;
    double dx, dy, dz, x, y, z, phase;
    hipfftHandle forward_plan, backward_plan;

    // Constant parameters
    const double lx = 1.0; // Length of the cubic lattice in x-direction
    const double ly = 1.0; // Length of the cubic lattice in y-direction
    const double lz = 1.0; // Length of the cubic lattice in z-direction

    // Wave vector components in reciprocal lattice units
    const double kx = 2.0 * M_PI / lx * 2; // Wave vector component in x-direction
    const double ky = 2.0 * M_PI / ly * 3; // Wave vector component in y-direction
    const double kz = 2.0 * M_PI / lz * 4; // Wave vector component in z-direction

    const int blockSize = 256;
    int numBlocks; 

    for (int n = 16; n <= 512; n *= 2) {

        nx = n;
        ny = n;
        nz = n;

        // Compute total number of lattice points
        nxyz = nx * ny * nz;

        // Compute distances between spatial lattice sites
        dx = lx / nx;
        dy = ly / ny;
        dz = lz / nz;

        // Allocate memory for wave, fft_3, d_dx, d_dy, and d_dz arrays on CPU
        hipDoubleComplex *wave = new hipDoubleComplex[nxyz];
        hipDoubleComplex *fft_3 = new hipDoubleComplex[nxyz];
        hipDoubleComplex *d_dx = new hipDoubleComplex[nxyz];
        hipDoubleComplex *d_dy = new hipDoubleComplex[nxyz];
        hipDoubleComplex *d_dz = new hipDoubleComplex[nxyz];

        // Allocate memory for wave, fft_3, d_dx, d_dy, and d_dz arrays on GPU
        hipDoubleComplex *wave_gpu;
        hipDoubleComplex *fft_3_gpu;
        hipDoubleComplex *d_dx_gpu;
        hipDoubleComplex *d_dy_gpu;
        hipDoubleComplex *d_dz_gpu;

        hipMalloc((void**)&wave_gpu, nxyz * sizeof(hipDoubleComplex));
        hipMalloc((void**)&fft_3_gpu, nxyz * sizeof(hipDoubleComplex));
        hipMalloc((void**)&d_dx_gpu, nxyz * sizeof(hipDoubleComplex));
        hipMalloc((void**)&d_dy_gpu, nxyz * sizeof(hipDoubleComplex));
        hipMalloc((void**)&d_dz_gpu, nxyz * sizeof(hipDoubleComplex));

        // Total time for all trials
        std::chrono::duration<double> total_time(0);

        for (int t = 1; t <= ntrials; ++t) {
            // Create cuFFT plans
            hipfftPlan3d(&forward_plan, nx, ny, nz, HIPFFT_Z2Z);
            hipfftPlan3d(&backward_plan, nx, ny, nz, HIPFFT_Z2Z);

            // Generate complex plane wave on the cubic lattice
            for (int i = 0; i < nxyz; ++i)
            {
                ix = i % nx;
                iy = (i / nx) % ny;
                iz = i / (nx * ny);
                x = (ix - nx / 2) * dx;
                y = (iy - ny / 2) * dy;
                z = (iz - nz / 2) * dz;
                phase = kx * x + ky * y + kz * z;

                wave[i] = make_hipDoubleComplex(cos(phase), sin(phase));
            }

            // For CUDA kernel to compute d/dx
            numBlocks = (nxyz + blockSize - 1) / blockSize;

            auto startTime = std::chrono::high_resolution_clock::now();

            // Copy data from CPU to GPU
            hipMemcpy(wave_gpu, wave, nxyz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

            // Perform forward FFT
            hipfftExecZ2Z(forward_plan, wave_gpu, fft_3_gpu, HIPFFT_FORWARD);

            // Make a copy of the forward transform
            hipMemcpy(wave_gpu, fft_3_gpu, nxyz * sizeof(hipDoubleComplex), hipMemcpyDeviceToDevice);

            // Compute d/dx
            computeDdx<<<numBlocks, blockSize>>>(wave_gpu, fft_3_gpu, kx, nxyz);
            hipfftExecZ2Z(backward_plan, fft_3_gpu, fft_3_gpu, HIPFFT_BACKWARD);
            computeScale<<<numBlocks, blockSize>>>(fft_3_gpu, d_dx_gpu, static_cast<double>(nxyz), nxyz);

            // Compute d/dy
            computeDdx<<<numBlocks, blockSize>>>(wave_gpu, fft_3_gpu, ky, nxyz);
            hipfftExecZ2Z(backward_plan, fft_3_gpu, fft_3_gpu, HIPFFT_BACKWARD);
            computeScale<<<numBlocks, blockSize>>>(fft_3_gpu, d_dy_gpu, static_cast<double>(nxyz), nxyz);

            // Compute d/dz
            computeDdx<<<numBlocks, blockSize>>>(wave_gpu, fft_3_gpu, kz, nxyz);
            hipfftExecZ2Z(backward_plan, fft_3_gpu, fft_3_gpu, HIPFFT_BACKWARD);
            computeScale<<<numBlocks, blockSize>>>(fft_3_gpu, d_dz_gpu, static_cast<double>(nxyz), nxyz);

            // Copy data from GPU to CPU
            hipMemcpy(d_dx, d_dx_gpu, nxyz * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
            hipMemcpy(d_dy, d_dy_gpu, nxyz * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
            hipMemcpy(d_dz, d_dz_gpu, nxyz * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

            auto endTime = std::chrono::high_resolution_clock::now();

            total_time += endTime - startTime;
        }

        // Free GPU memory
        hipFree(wave_gpu);
        hipFree(fft_3_gpu);
        hipFree(d_dx_gpu);
        hipFree(d_dy_gpu);
        hipFree(d_dz_gpu);

        // Destroy cuFFT plans
        hipfftDestroy(forward_plan);
        hipfftDestroy(backward_plan);

        // Free CPU memory
        delete[] wave;
        delete[] fft_3;
        delete[] d_dx;
        delete[] d_dy;
        delete[] d_dz;

        double avg_time = total_time.count() / ntrials;

        std::cout << "n=" << n << ": " << avg_time << std::endl;
    }

    return 0;
}
