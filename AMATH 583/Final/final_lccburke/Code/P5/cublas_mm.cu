
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>
#include <hipblas.h>

// Perform matrix multiplication using cuBLAS
void cublas_matrix_multiply(int n, const double* A, const double* B, double* C, double alpha, double beta) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Use cublasDgemm function for matrix multiplication
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n, &beta, C, n);

    hipblasDestroy(handle);
}

int main() {
    const double alpha = 1.0, beta = 0.0;
    const int n_start = 16, n_end = 8192, n_step = 2;

    double *d_A, *d_B, *d_C;

    for (int n = n_start; n <= n_end; n *= n_step) {
        std::vector<double> A(n * n, 1.0);
        std::vector<double> B(n * n, 1.0);
        std::vector<double> C(n * n, 0.0);

        // Allocate device memory
        hipMalloc((void**)&d_A, n * n * sizeof(double));
        hipMalloc((void**)&d_B, n * n * sizeof(double));
        hipMalloc((void**)&d_C, n * n * sizeof(double));

        // Copy vectors from host to device
        hipMemcpy(d_A, A.data(), n * n * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B.data(), n * n * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_C, C.data(), n * n * sizeof(double), hipMemcpyHostToDevice);

        const int n_trials = 3;
        double total_time = 0.0;

        // Perform multiplication n_trials times and measure average execution time
        for (int trial = 0; trial < n_trials; ++trial) {
            auto start_time = std::chrono::high_resolution_clock::now();
            cublas_matrix_multiply(n, d_A, d_B, d_C, alpha, beta);
            hipDeviceSynchronize(); // Ensure computation is done
            auto end_time = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double> diff = end_time - start_time;

            total_time += diff.count();
        }

        // Copy array back to host
        hipMemcpy(C.data(), d_C, n * n * sizeof(double), hipMemcpyDeviceToHost);

        double avg_time = total_time / n_trials;
        // Performance in GFLOPs = (2n^3) / (avg_time * 10^9)
        double gflops = (2.0 * n * n * n) / (avg_time * 1e9);

        std::cout << "Matrix Size: " << n << " Avg Time: " << avg_time << " GFLOPs: " << gflops << std::endl;

        // Cleanup
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    return 0;
}
