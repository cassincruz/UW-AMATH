#include <iostream>
#include <hip/hip_runtime.h>

#include <chrono>

int main()
{
    std::cout << "Time to/from CPU/GPU by buffer size" << std::endl;
    for (int dataSize = 8; dataSize <= 256; dataSize *= 2) {

        std::cout<< dataSize << " bytes: ";
        
        // Allocate and initialize input data on the host
        int *hostData = new int[dataSize]; 
        for (int i = 0; i < dataSize; i++) {
            hostData[i] = i + 1;
        }

        // Allocate memory on the device
        int *deviceData;
        hipMalloc((void **)&deviceData, dataSize * sizeof(int));

        // Copy input data from host to device
        std::chrono::high_resolution_clock::time_point startTime = std::chrono::high_resolution_clock::now();
        hipMemcpy(deviceData, hostData, dataSize * sizeof(int), hipMemcpyHostToDevice);
        std::chrono::high_resolution_clock::time_point endTime = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();

        std::cout << "To GPU: " << duration << " microseconds "; 

        // Copy data back to host from device
        startTime = std::chrono::high_resolution_clock::now();
        hipMemcpy(hostData, deviceData, dataSize * sizeof(int), hipMemcpyDeviceToHost);
        endTime = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::microseconds>(endTime - startTime).count();

        std::cout << "To CPU: " << duration << " microseconds" << std::endl;

        // Free memory
        delete[] hostData;
        hipFree(deviceData);
    }

    return 0;
}
